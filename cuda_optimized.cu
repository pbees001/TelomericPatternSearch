
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <cstring>
#include <time.h>
#include <vector>
#include <chrono>
#include <thread>
using namespace std;
//cuda_opt2 version - Final
#define maxPatternLength 1300000//Increase this limit if max read length of DNA sequence exceed this value in the dataset
#define maxEntries 1024
#define maxThreads 1024
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

__global__ void kernel(char *d_c, int* d_match, char *d_pat, int *d_lps, int *d_numOfEntries, int *d_maxPattLength){
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    int maxPattlength = *d_maxPattLength;
    int n = *d_numOfEntries;
    char pat[] = "TTAGGGTTAGGGTTAGGGTTAGGG";
    if(idx < n){
        int M = 24;
        int N = 0;
        while(d_c[idx * maxPattlength + N]!='\0')
            N++;
        int pattern_found_count = 0;
		int rearEndRange = 20000;

        int i = 0, j =0;
        int lastMatchedIndex = -1;
        if(d_match[idx]!=0)
            return;
        while (i < N) {
            if (pat[j] == d_c[idx * maxPattlength + i]) {
                j++;
                i++;
            }
            if (j == M) {
                j = 0;
                pattern_found_count+=1;
                lastMatchedIndex = i;
            }
            else if (i < N && pat[j] != d_c[idx * maxPattlength + i]) {
                if (j != 0)
                    j = d_lps[j-1];
                else
                    i = i + 1;
            }
        }
        if(pattern_found_count > 1 && i-lastMatchedIndex < rearEndRange){
            d_match[idx]=1;
        }

    }
}

void computeLPSArray(char* pat, int M, int* lps)
{
    int len = 0;
    lps[0] = 0;
    int i = 1;
    while (i < M) {
        if (pat[i] == pat[len]) {
            len++;
            lps[i] = len;
            i++;
        }
        else
        {
            if (len != 0) {
                len = lps[len - 1];
            }
            else
            {
                lps[i] = 0;
                i++;
            }
        }
    }
}

int main( int argc, char **argv ){
    if( argc <= 2 ){
        cerr << "Usage is: "<<argv[0]<<" [infilename] [outfilename]" << std::endl;
        return -1;
    }

    ifstream input(argv[1]);
    ofstream output(argv[2]);
    if(!input.good()){
        cerr << "Error opening file "<<argv[1] << std::endl;
        return -1;
    }

    clock_t t0, t1, t2;
    double t1sum=0.0;
    double t2sum=0.0;
    double dataLoadTime = 0.0, kernelComputionTime = 0.0;

    int N = 0, maxPattLength = maxPatternLength;
    int total_threadblocks = 0, total_threads = 0;
    string line;

    //fetching total no.of DNA pieces being processed
    while( getline( input, line ).good() ){
        if( line[0] == '>' ){
            N++;
        }
    }
    printf("Total DNA pieces in dataset : %d\n",N);

    t0 = clock();//Starting the data initialization time

    if(N > maxEntries)
        N = maxEntries;//limiting data allocation to N DNA pieces
    int j=0;
    int *match = new int[N];//flag array to segragate matched sequences
    for(int i=0;i<N;i++){
        match[i]= -1;
    }
    int pattLength = 24;
    char pattern[] = "TTAGGGTTAGGGTTAGGGTTAGGG";
    int *lps = new int[pattLength];
    computeLPSArray(pattern,pattLength,lps);//computing KMP hash map

    char (*seqs)[maxPatternLength] = new char [N][maxPatternLength];//DNA sequences array
    string names[N];//readID array
    char *d_c,*d_pat;
    int *d_match,*d_lps;
    int *d_numOfEntries, *d_maxPattLength;

    hipMalloc((void**)&d_pat,pattLength*sizeof(char));
    hipMalloc((void**)&d_lps,pattLength*sizeof(int));
    hipMalloc((void**)&d_numOfEntries,sizeof(int));
    hipMalloc((void**)&d_maxPattLength,sizeof(int));
    hipMalloc((void**)&d_match,N * sizeof(int));//device memory allocation for flag array
    hipHostMalloc((int**)&match,N * sizeof(int), hipHostMallocDefault);//pinned memory
    hipMalloc((void**)&d_c,N*maxPatternLength*sizeof(char));//device memory allocation for DNA sequences
    hipHostMalloc((char**)&seqs,N*maxPatternLength*sizeof(char), hipHostMallocDefault);//pinned memory
    cudaCheckErrors("hipMalloc failure");

    hipMemcpy(d_pat, pattern, pattLength*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_lps, lps, pattLength*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_numOfEntries, &N, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_maxPattLength, &maxPattLength, sizeof(int), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy failure");

    string name, content,tempString = "";
    input.clear();
    input.seekg(0);
    while( std::getline( input, line ).good() ){
        if( line.empty() || line[0] == '>' ){
            if( !name.empty() ){
                strcpy(seqs[j],content.c_str());
                names[j]=name;
                match[j]=0;
                j++;
                name.clear();

                if(j == N){//processing N DNA pieces at a time
                    hipMemcpy(d_match, match, N*sizeof(int), hipMemcpyHostToDevice);
                    hipMemcpy(d_c, &seqs[0][0], N*maxPatternLength*sizeof(char), hipMemcpyHostToDevice);
                    cudaCheckErrors("hipMemcpy H2D failure");

                    t1 = clock();//starting kernel computation time
                    t1sum = ((double)(t1-t0))/CLOCKS_PER_SEC;
                    dataLoadTime+=t1sum;

                    //Launching kernel for N DNA sequences KMP patten search processing
                    kernel <<< (N+maxThreads-1)/maxThreads,maxThreads >>> (d_c, d_match, d_pat, d_lps, d_numOfEntries, d_maxPattLength);
                    cudaCheckErrors("kernel launch failure");
                    total_threads+= maxThreads;//calculating total threads used
                    total_threadblocks+= (N+maxThreads-1)/maxThreads;//calculating total thread blocks used


                    t2 = clock();//calculating kernel computation end time
                    t2sum = ((double)(t2-t1))/CLOCKS_PER_SEC;
                    kernelComputionTime+=t2sum;

                    hipMemcpy(match, d_match, N*sizeof(int), hipMemcpyDeviceToHost);//write back device memory to host memory, flag array to find matched DNA sequences
                    cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");

                    //writing output to file
                    for(int k=0;k<N;k++){
                        if(match[k]== 1){
                            output<<'>'<<names[k]<<'\n';
                            output<<seqs[k]<<'\n';
                        }
                        match[k]= -1;
                        names[k].clear();
                        strcpy(seqs[k],tempString.c_str());
                    }
                    j = 0;
                    t0 = clock();
                }
            }
            if( !line.empty() ){
                name = line.substr(1);
            }
            content.clear();
        } else if( !name.empty() ){
            if( line.find(' ') != std::string::npos ){
                name.clear();
                content.clear();
            } else {
                content += line;
            }
        }
    }
    if( !name.empty() ){
        strcpy(seqs[j],content.c_str());
        names[j] = name;
        match[j] = 0;
        j++;
    }

    hipMemcpy(d_match, match, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_c, &seqs[0][0], N*maxPatternLength*sizeof(char), hipMemcpyHostToDevice);//copying DNA seqs from host to device
    cudaCheckErrors("hipMemcpy H2D failure");

    t1 = clock();
    t1sum = ((double)(t1-t0))/CLOCKS_PER_SEC;
    dataLoadTime+=t1sum;//calculating data initialization time

    //final kernel launch
    kernel <<< (N+maxThreads-1)/maxThreads,maxThreads >>> (d_c, d_match, d_pat, d_lps, d_numOfEntries, d_maxPattLength);
    cudaCheckErrors("kernel execution failure");
    total_threads+= maxThreads;
    total_threadblocks+= (N+maxThreads-1)/maxThreads;

    t2 = clock();
    t2sum = ((double)(t2-t1))/CLOCKS_PER_SEC;
    kernelComputionTime+=t2sum;
    hipMemcpy(match, d_match, N*sizeof(int), hipMemcpyDeviceToHost);
    cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");

    for(int i=0;i<N;i++){
        if(match[i]==1){
            output<<'>'<<names[i]<<'\n';
            output<<seqs[i]<<'\n';
        }
    }

    printf("Overall Data Initialization time: %f seconds\n",dataLoadTime);
    printf("Overall Kernel computation time: %f seconds\n",kernelComputionTime);
    printf("Total Threads used: %d\n", total_threads);
    printf("Total Thread-Blocks used: %d\n", total_threadblocks);
    input.close();
    output.close();

    hipFree(d_c);
    hipFree(d_pat);
    hipFree(d_match);
    hipFree(d_numOfEntries);
    cudaCheckErrors("Free memory failure");

    match = NULL;
    delete[] match;

    return 0;
}
